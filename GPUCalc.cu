#include "hip/hip_runtime.h"
﻿#include "GPUCalc.cuh"
#include "hsvrgb_GPU.cuh"

//Converts a pixel's coordinates to a complex number, which will be stored in result
__device__
void coordsToComplexGPU(const int* x, const int* y, const fracState* state, hipDoubleComplex* result)
{
    result->x = ((double)*x + state->xPanOffset) / state->xZoomScale;
    result->y = ((double)*y + state->yPanOffset) / state->yZoomScale;
}

/*Calculates the number of iterations required to determine whether the passed complex number is 
in the mandelbrot set or not and calculates a smooth color based on the number of iterations. 
The result will be placed in the passed double* smooth and will be -1.0 if the pixel is in the 
mandelbrot set and should be colored black*/
__device__
void getSmoothColorGPU(const hipDoubleComplex* complexNum, double* smooth)
{
    int iters = 0;
    hipDoubleComplex z = make_hipDoubleComplex(0, 0);
    /*(z.y * z.y) + (z.x * z.x) <= 4 is equivalent to abs(z) <= 2
    While the former may look more complicated, it is about twice as efficient as the latter*/
    for (; (z.y * z.y) + (z.x * z.x) <= 4 && iters < MAX_ITER; (iters)++)
    {
        z = hipCmul(z, z);
        z = hipCadd(z, *complexNum);
    }
    iters == MAX_ITER ? *smooth = -1.0 : calcSmoothColorGPU(&z, &iters, smooth);
}

//Uses the mandelbrot smooth-coloring algorithm (https://stackoverflow.com/questions/369438/smooth-spectrum-for-mandelbrot-set-rendering)
//to calculate a value between 0 and 1 which will be used to determine the color of a pixel
__device__
void calcSmoothColorGPU(const hipDoubleComplex* complexNum, const int* iters, double* smooth)
{
    //sqrt((complexNum->x * complexNum->x) + (complexNum->y * complexNum->y))
    //is much faster than abs(z) but gives the same result
    double complexAbs = sqrt((complexNum->x * complexNum->x) + (complexNum->y * complexNum->y));
    double complexDoubleLog = log10(log10(complexAbs));
    *smooth = *iters + 1 - (complexDoubleLog / log10(2.0));
}

//Calculates results for a section of the fractal
__global__
void makeFracGPU(RGB* resultArr, const fracState* state)
{
    double smooth;
    int sectionHeight = (int)floor((double)(WINDOW_HEIGHT / gridDim.x));
    int sectionWidth = (int)floor((double)(WINDOW_WIDTH / blockDim.x));
    float h, s = 0.7, v = 1.0, r, g, b;
    hipDoubleComplex complexPixel = make_hipDoubleComplex(0, 0);

    for (int y = blockIdx.x * sectionHeight; y < (blockIdx.x + 1) * sectionHeight; y++)
    {
        for (int x = threadIdx.x * sectionWidth; x < (threadIdx.x + 1) * sectionWidth; x++)
        {
            coordsToComplexGPU(&x, &y, state, &complexPixel);
            getSmoothColorGPU(&complexPixel, &smooth);

            if (smooth == -1.0)
                r = g = b = 0;
            else
            {
                h = smooth + 255;
                HSVtoRGB_GPU(r, g, b, h, s, v);
            }

            resultArr[y * WINDOW_WIDTH + x].r = r * 255;
            resultArr[y * WINDOW_WIDTH + x].g = g * 255;
            resultArr[y * WINDOW_WIDTH + x].b = b * 255;
        }
    }
    return;
}